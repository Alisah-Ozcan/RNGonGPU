#include "hip/hip_runtime.h"
// Copyright 2025 Alişah Özcan
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0

#include "aes_rng.cuh"
#include "base_rng.cuh"
#include <random>

namespace rngongpu
{
    void BaseRNG_AES::init() {
        this -> seed = new Data32[8];

        std::random_device rd;
        std::mt19937_64 gen(rd());
        std::uniform_int_distribution<Data32> dist(0, std::numeric_limits<Data32>::max());
        
        for (int i = 0; i < 8; i++) this -> seed[i] = dist(gen);

        // Results of Block_Encrypt(0, 1) and Block_Encrypt(0, 2)
        Data32 temp[8];
        temp[0] = 0x58E2FCCE;
        temp[1] = 0xFA7E3061;
        temp[2] = 0x367F1D57;
        temp[3] = 0xA4E7455A;
        temp[4] = 0x0388DACE;
        temp[5] = 0x60B6A392;
        temp[6] = 0xF328C2B9;
        temp[7] = 0x71B2FE78;

        this -> key = new Data32[4];
        this -> nonce = new Data32[4];

        for (int i = 0; i < 4; i++) {
            this -> key[i] = temp[i] ^ seed[i];
            this -> nonce[i] = temp[i] ^ seed[i+4];
        }

        // Allocate RCON values
        RNGONGPU_CUDA_CHECK(hipMallocManaged(&(this -> rcon), RCON_SIZE * sizeof(Data32)));
        for (int i = 0; i < RCON_SIZE; i++) {
            this -> rcon[i] = RCON32[i];
        }

        // Allocate Tables
        RNGONGPU_CUDA_CHECK(hipMallocManaged(&(this -> t0), TABLE_SIZE * sizeof(Data32)));
        RNGONGPU_CUDA_CHECK(hipMallocManaged(&(this -> t1), TABLE_SIZE * sizeof(Data32)));
        RNGONGPU_CUDA_CHECK(hipMallocManaged(&(this -> t2), TABLE_SIZE * sizeof(Data32)));
        RNGONGPU_CUDA_CHECK(hipMallocManaged(&(this -> t3), TABLE_SIZE * sizeof(Data32)));
        RNGONGPU_CUDA_CHECK(hipMallocManaged(&(this -> t4), TABLE_SIZE * sizeof(Data32)));
        RNGONGPU_CUDA_CHECK(hipMallocManaged(&(this -> t4_0), TABLE_SIZE * sizeof(Data32)));
        RNGONGPU_CUDA_CHECK(hipMallocManaged(&(this -> t4_1), TABLE_SIZE * sizeof(Data32)));
        RNGONGPU_CUDA_CHECK(hipMallocManaged(&(this -> t4_2), TABLE_SIZE * sizeof(Data32)));
        RNGONGPU_CUDA_CHECK(hipMallocManaged(&(this -> t4_3), TABLE_SIZE * sizeof(Data32)));
        RNGONGPU_CUDA_CHECK(hipMallocManaged(&(this -> SAES_d), 256 * sizeof(Data8))); // Cihangir
        for (int i = 0; i < TABLE_SIZE; i++) {
            this -> t0[i] = T0[i];
            this -> t1[i] = T1[i];
            this -> t2[i] = T2[i];
            this -> t3[i] = T3[i];
            this -> t4[i] = T4[i];
            this -> t4_0[i] = T4_0[i];
            this -> t4_1[i] = T4_1[i];
            this -> t4_2[i] = T4_2[i];
            this -> t4_3[i] = T4_3[i];
        }
        for (int i = 0; i < 256; i++) this -> SAES_d[i] = SAES[i]; // Cihangir

        RNGONGPU_CUDA_CHECK(hipMallocManaged(&(this -> roundKeys), AES_128_KEY_SIZE_INT * sizeof(Data32)));
        
        hipMalloc(&(this -> d_nonce), 4 * sizeof(Data32));
        hipMemcpy(this -> d_nonce, this -> nonce, 4 * sizeof(Data32), hipMemcpyHostToDevice);

        // Key expansion
        keyExpansion(this -> key, this -> roundKeys);
        initState();
    }
    void BaseRNG_AES::initState() {}
    void BaseRNG_AES::increment_nonce(Data32 N) {
        if (this -> nonce[3] + N < this -> nonce[3]) {
            this -> nonce[2] += 1;
        }
        this -> nonce[3] += N;
        
        hipMemcpy(this -> d_nonce, this -> nonce, 4 * sizeof(Data32), hipMemcpyHostToDevice);
    }

    // generate random bits on the device. Write N bytes to res 
    // using BLOCKS blocks with THREADS threads each.
    void BaseRNG_AES::gen_random_bytes(int N, int BLOCKS, int THREADS, Data64* res) {
        int num_u64 = (N + 7) / 8;
        // Calculate the range for each thread
        Data64* range;
        RNGONGPU_CUDA_CHECK(hipMallocManaged(&range, sizeof(Data64)));
        int threadCount = BLOCKS * THREADS;
        double threadCount_d = (double) num_u64;
        double threadRange = threadCount_d / (threadCount * 2);
        *range = ceil(threadRange);

        printf("N: %u, range: %llu, BLOCKS: %u, THREADS: %u\n", num_u64, *range, BLOCKS, THREADS);
        printf("Calling kernel to generate %u numbers, range: %llu\n", num_u64, *range);
        counterWithOneTableExtendedSharedMemoryBytePermPartlyExtendedSBoxCihangir<<<BLOCKS, THREADS>>>(this -> d_nonce, this -> roundKeys, this -> t0, this -> t4, range, this -> SAES_d, res, num_u64);
        hipDeviceSynchronize();
        //printLastCUDAError();

        // Free alocated arrays
        hipFree(range);

        this -> increment_nonce(num_u64 + 1 / 2);
    }
    BaseRNG_AES::BaseRNG_AES() : seed(nullptr), nonce(nullptr), key(nullptr) {this -> init();}

    // tune the object for desired output in the next function call 
    // ex: set the stddev and mean for Normal distribution objects
    // virtual void set_state() = 0;

    // virtual void gen_random_u32(int N, u32* res) = 0;
    // virtual void gen_random_u32_mod_p(int N, u32 p, u32* res) = 0;
    // virtual void gen_random_u32_mod_p(int N, u32* p, u32 p_num, u32* res) = 0;
    // virtual void gen_random_u64(int N, u64* res) = 0;
    // virtual void gen_random_u64_mod_p(int N, u64 p, u64* res) = 0;
    // virtual void gen_random_u64_mod_p(int N, u64* p, u32 p_num, u64* res) = 0;
    // virtual void gen_random_f32(int N, f32* res) = 0;
    // virtual void gen_random_f64(int N, f64* res) = 0;

    BaseRNG_AES::~BaseRNG_AES() {
        hipFree(this -> t0);
        hipFree(this -> t1);
        hipFree(this -> t2);
        hipFree(this -> t3);
        hipFree(this -> t4);
        hipFree(this -> t4_0);
        hipFree(this -> t4_1);
        hipFree(this -> t4_2);
        hipFree(this -> t4_3);
        hipFree(this -> rcon);
        hipFree(this -> SAES_d);
        hipFree(this -> d_nonce);
        hipFree(this -> roundKeys);
    }
} // namespace rngongpu