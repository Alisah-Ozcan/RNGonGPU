#include "hip/hip_runtime.h"
// Copyright 2025 Alişah Özcan
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0

#include "aes_rng.cuh"
#include "base_rng.cuh"
#include <random>

namespace rngongpu
{
    void test_aes() {
        std::cout << "TEST SUCCESS\n";
    }
    void BaseRNG_AES::init() {
        this -> seed = new Data32[4];
        this -> nonce = new Data32[4];
        std::random_device rd;
        std::mt19937_64 gen(rd());
        std::uniform_int_distribution<Data32> dist(0, std::numeric_limits<Data32>::max());
        
        this -> seed[0] = dist(gen);
        this -> seed[1] = dist(gen);
        this -> seed[2] = dist(gen);
        this -> seed[3] = dist(gen); 

        this -> nonce[0] = dist(gen);
        this -> nonce[1] = dist(gen);
        this -> nonce[2] = dist(gen);
        this -> nonce[3] = dist(gen);

        // this -> nonce[0] = 0x3243F6A8U;
        // this -> nonce[1] = 0x885A308DU;
        // this -> nonce[2] = 0x313198A2U;
        // this -> nonce[3] = 0x00000000U;

        // this -> seed[0] = 0x2B7E1516U;
        // this -> seed[1] = 0x28AED2A6U;
        // this -> seed[2] = 0xABF71588U;
        // this -> seed[3] = 0x09CF4F3CU;

        // Allocate RCON values
        RNGONGPU_CUDA_CHECK(hipMallocManaged(&(this -> rcon), RCON_SIZE * sizeof(Data32)));
        for (int i = 0; i < RCON_SIZE; i++) {
            this -> rcon[i] = RCON32[i];
        }

        // Allocate Tables
        RNGONGPU_CUDA_CHECK(hipMallocManaged(&(this -> t0), TABLE_SIZE * sizeof(Data32)));
        RNGONGPU_CUDA_CHECK(hipMallocManaged(&(this -> t1), TABLE_SIZE * sizeof(Data32)));
        RNGONGPU_CUDA_CHECK(hipMallocManaged(&(this -> t2), TABLE_SIZE * sizeof(Data32)));
        RNGONGPU_CUDA_CHECK(hipMallocManaged(&(this -> t3), TABLE_SIZE * sizeof(Data32)));
        RNGONGPU_CUDA_CHECK(hipMallocManaged(&(this -> t4), TABLE_SIZE * sizeof(Data32)));
        RNGONGPU_CUDA_CHECK(hipMallocManaged(&(this -> t4_0), TABLE_SIZE * sizeof(Data32)));
        RNGONGPU_CUDA_CHECK(hipMallocManaged(&(this -> t4_1), TABLE_SIZE * sizeof(Data32)));
        RNGONGPU_CUDA_CHECK(hipMallocManaged(&(this -> t4_2), TABLE_SIZE * sizeof(Data32)));
        RNGONGPU_CUDA_CHECK(hipMallocManaged(&(this -> t4_3), TABLE_SIZE * sizeof(Data32)));
        RNGONGPU_CUDA_CHECK(hipMallocManaged(&(this -> SAES_d), 256 * sizeof(Data8))); // Cihangir
        for (int i = 0; i < TABLE_SIZE; i++) {
            this -> t0[i] = T0[i];
            this -> t1[i] = T1[i];
            this -> t2[i] = T2[i];
            this -> t3[i] = T3[i];
            this -> t4[i] = T4[i];
            this -> t4_0[i] = T4_0[i];
            this -> t4_1[i] = T4_1[i];
            this -> t4_2[i] = T4_2[i];
            this -> t4_3[i] = T4_3[i];
        }
        for (int i = 0; i < 256; i++) this -> SAES_d[i] = SAES[i]; // Cihangir

        RNGONGPU_CUDA_CHECK(hipMallocManaged(&(this -> roundKeys), AES_128_KEY_SIZE_INT * sizeof(Data32)));
        
        hipMalloc(&(this -> d_nonce), 4 * sizeof(Data32));
        hipMemcpy(this -> d_nonce, this -> nonce, 4 * sizeof(Data32), hipMemcpyHostToDevice);

        // Key expansion
        keyExpansion(this -> seed, this -> roundKeys);
        initState();
    }
    void BaseRNG_AES::initState() {}
    void BaseRNG_AES::increment_nonce(Data32 N) {
        if (this -> nonce[3] + N < this -> nonce[3]) {
            this -> nonce[2] += 1;
        }
        this -> nonce[3] += N;
        
        hipMemcpy(this -> d_nonce, this -> nonce, 4 * sizeof(Data32), hipMemcpyHostToDevice);
    }

    // generate random bits on the device. Write N bytes to res 
    // using BLOCKS blocks with THREADS threads each.
    void BaseRNG_AES::gen_random_bytes(int N, int BLOCKS, int THREADS, Data64* res) {
        int num_u64 = (N + 7) / 8;
        // Calculate the range for each thread
        Data64* range;
        RNGONGPU_CUDA_CHECK(hipMallocManaged(&range, sizeof(Data64)));
        int threadCount = BLOCKS * THREADS;
        double threadCount_d = (double) num_u64;
        double threadRange = threadCount_d / (threadCount * 2);
        *range = ceil(threadRange);

        printf("N: %u, range: %llu, BLOCKS: %u, THREADS: %u\n", num_u64, *range, BLOCKS, THREADS);
        printf("Calling kernel to generate %u numbers, range: %llu\n", num_u64, *range);
        counterWithOneTableExtendedSharedMemoryBytePermPartlyExtendedSBoxCihangir<<<BLOCKS, THREADS>>>(this -> d_nonce, this -> roundKeys, this -> t0, this -> t4, range, this -> SAES_d, res, num_u64);
        hipDeviceSynchronize();
        //printLastCUDAError();

        // Free alocated arrays
        hipFree(range);

        this -> increment_nonce(num_u64 + 1 / 2);
    }
    BaseRNG_AES::BaseRNG_AES() : seed(nullptr), nonce(nullptr) {this -> init();}

    // tune the object for desired output in the next function call 
    // ex: set the stddev and mean for Normal distribution objects
    // virtual void set_state() = 0;

    // virtual void gen_random_u32(int N, u32* res) = 0;
    // virtual void gen_random_u32_mod_p(int N, u32 p, u32* res) = 0;
    // virtual void gen_random_u32_mod_p(int N, u32* p, u32 p_num, u32* res) = 0;
    // virtual void gen_random_u64(int N, u64* res) = 0;
    // virtual void gen_random_u64_mod_p(int N, u64 p, u64* res) = 0;
    // virtual void gen_random_u64_mod_p(int N, u64* p, u32 p_num, u64* res) = 0;
    // virtual void gen_random_f32(int N, f32* res) = 0;
    // virtual void gen_random_f64(int N, f64* res) = 0;

    BaseRNG_AES::~BaseRNG_AES() {
        hipFree(this -> t0);
        hipFree(this -> t1);
        hipFree(this -> t2);
        hipFree(this -> t3);
        hipFree(this -> t4);
        hipFree(this -> t4_0);
        hipFree(this -> t4_1);
        hipFree(this -> t4_2);
        hipFree(this -> t4_3);
        hipFree(this -> rcon);
        hipFree(this -> SAES_d);
        hipFree(this -> d_nonce);
        hipFree(this -> roundKeys);
    }
} // namespace rngongpu